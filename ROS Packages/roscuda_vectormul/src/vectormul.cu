#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>


__global__ void add(int *a, int *b, int *c)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

void testmain(int size, int *c)
{
  int *a, *b; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c;
  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); a[0]=1;
  b = (int *)malloc(size); b[0]=4;
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU
  add<<<1,1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  // Cleanup
  free(a); free(b);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return;
}
